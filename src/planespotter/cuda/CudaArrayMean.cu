#include "hip/hip_runtime.h"
extern "C"

__global__ void mean(int len, float *a, float *b, float *mean) {

    int block = blockIdx.x + blockDim.x + threadIdx.x;
    int max = i + len;

    int i;
    for (i = block; i < max; i++) {
        mean[i] = a[i] + b[i];
    }

}